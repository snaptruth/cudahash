#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_occupancy.h>
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cudasha256.h"
#include "cuda_helper.h"

__constant__ static unsigned int __align__(8) c_midstate76[8];
__constant__ static unsigned int __align__(8) c_dataEnd80[4];

const __constant__  unsigned int __align__(8) c_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};
__constant__ static unsigned int __align__(8) c_K[64];
__constant__ static unsigned int __align__(8) c_target[2];

static unsigned int* d_resNonces[MAX_GPUS] = { 0 };
volatile unsigned int *pFinished = NULL;

unsigned int uiBlocksPerGrid = 0;
unsigned int uiThreadsPerBlock = 0;
// ------------------------------------------------------------------------------------------------

static const unsigned int cpu_H256[8] = {
	0x6A09E667U, 0xBB67AE85U, 0x3C6EF372U, 0xA54FF53AU,
	0x510E527FU, 0x9B05688CU, 0x1F83D9ABU, 0x5BE0CD19U
};

static const unsigned int cpu_K[64] = {
	0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
	0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3, 0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
	0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
	0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
	0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13, 0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
	0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
	0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
	0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208, 0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2
};

#define ROTR ROTR32

__host__
static void sha256_step1_host(unsigned int a, unsigned int b, unsigned int c, unsigned int &d,
	unsigned int e, unsigned int f, unsigned int g, unsigned int &h,
	unsigned int in, const unsigned int Kshared)
{
	unsigned int t1,t2;
	unsigned int vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	unsigned int bsg21 = ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
	unsigned int bsg20 = ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
	unsigned int andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__host__
static void sha256_step2_host(unsigned int a, unsigned int b, unsigned int c, unsigned int &d,
	unsigned int e, unsigned int f, unsigned int g, unsigned int &h,
	unsigned int* in, unsigned int pc, const unsigned int Kshared)
{
	unsigned int t1,t2;

	int pcidx1 = (pc-2)  & 0xF;
	int pcidx2 = (pc-7)  & 0xF;
	int pcidx3 = (pc-15) & 0xF;

	unsigned int inx0 = in[pc];
	unsigned int inx1 = in[pcidx1];
	unsigned int inx2 = in[pcidx2];
	unsigned int inx3 = in[pcidx3];

	unsigned int ssg21 = ROTR(inx1, 17) ^ ROTR(inx1, 19) ^ SPH_T32((inx1) >> 10); //ssg2_1(inx1);
	unsigned int ssg20 = ROTR(inx3, 7) ^ ROTR(inx3, 18) ^ SPH_T32((inx3) >> 3); //ssg2_0(inx3);
	unsigned int vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	unsigned int bsg21 = ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
	unsigned int bsg20 = ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
	unsigned int andorv = ((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}

__host__
static void sha256_round_body_host(unsigned int* in, unsigned int* state, const unsigned int* Kshared)
{
	unsigned int a = state[0];
	unsigned int b = state[1];
	unsigned int c = state[2];
	unsigned int d = state[3];
	unsigned int e = state[4];
	unsigned int f = state[5];
	unsigned int g = state[6];
	unsigned int h = state[7];

	sha256_step1_host(a,b,c,d,e,f,g,h,in[ 0], Kshared[ 0]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[ 1], Kshared[ 1]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[ 2], Kshared[ 2]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[ 3], Kshared[ 3]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[ 4], Kshared[ 4]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[ 5], Kshared[ 5]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[ 6], Kshared[ 6]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[ 7], Kshared[ 7]);
	sha256_step1_host(a,b,c,d,e,f,g,h,in[ 8], Kshared[ 8]);
	sha256_step1_host(h,a,b,c,d,e,f,g,in[ 9], Kshared[ 9]);
	sha256_step1_host(g,h,a,b,c,d,e,f,in[10], Kshared[10]);
	sha256_step1_host(f,g,h,a,b,c,d,e,in[11], Kshared[11]);
	sha256_step1_host(e,f,g,h,a,b,c,d,in[12], Kshared[12]);
	sha256_step1_host(d,e,f,g,h,a,b,c,in[13], Kshared[13]);
	sha256_step1_host(c,d,e,f,g,h,a,b,in[14], Kshared[14]);
	sha256_step1_host(b,c,d,e,f,g,h,a,in[15], Kshared[15]);

	for (int i=0; i<3; i++)
	{
		sha256_step2_host(a,b,c,d,e,f,g,h,in,0, Kshared[16+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,1, Kshared[17+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,2, Kshared[18+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,3, Kshared[19+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,4, Kshared[20+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,5, Kshared[21+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,6, Kshared[22+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,7, Kshared[23+16*i]);
		sha256_step2_host(a,b,c,d,e,f,g,h,in,8, Kshared[24+16*i]);
		sha256_step2_host(h,a,b,c,d,e,f,g,in,9, Kshared[25+16*i]);
		sha256_step2_host(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha256_step2_host(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha256_step2_host(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha256_step2_host(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha256_step2_host(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha256_step2_host(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}

#define xor3b(a,b,c) (a ^ b ^ c)

__device__ __forceinline__ unsigned int bsg2_0(const unsigned int x)
{
	return xor3b(ROTR32(x,2),ROTR32(x,13),ROTR32(x,22));
}

__device__ __forceinline__ unsigned int bsg2_1(const unsigned int x)
{
	return xor3b(ROTR32(x,6),ROTR32(x,11),ROTR32(x,25));
}

__device__ __forceinline__ unsigned int ssg2_0(const unsigned int x)
{
	return xor3b(ROTR32(x,7),ROTR32(x,18),(x>>3));
}

__device__ __forceinline__ unsigned int ssg2_1(const unsigned int x)
{
	return xor3b(ROTR32(x,17),ROTR32(x,19),(x>>10));
}

__device__ __forceinline__ unsigned int andor32(const unsigned int a, const unsigned int b, const unsigned int c)
{
	unsigned int result;
	asm("{\n\t"
		".reg .u32 m,n,o;\n\t"
		"and.b32 m,  %1, %2;\n\t"
		" or.b32 n,  %1, %2;\n\t"
		"and.b32 o,   n, %3;\n\t"
		" or.b32 %0,  m, o ;\n\t"
		"}\n\t" : "=r"(result) : "r"(a), "r"(b), "r"(c)
	);
	return result;
}

__device__ __forceinline__ uint2 vectorizeswap(uint64_t v) {
	uint2 result;
	asm("mov.b64 {%0,%1},%2; \n\t"
		: "=r"(result.y), "=r"(result.x) : "l"(v));
	return result;
}

__device__
static void sha2_step1(unsigned int a, unsigned int b, unsigned int c, unsigned int &d, unsigned int e, unsigned int f, unsigned int g, unsigned int &h,
	unsigned int in, const unsigned int Kshared)
{
	unsigned int t1,t2;
	unsigned int vxandx = xandx(e, f, g);
	unsigned int bsg21 = bsg2_1(e);
	unsigned int bsg20 = bsg2_0(a);
	unsigned int andorv = andor32(a,b,c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

__device__
static void sha2_step2(unsigned int a, unsigned int b, unsigned int c, unsigned int &d, unsigned int e, unsigned int f, unsigned int g, unsigned int &h,
	unsigned int* in, unsigned int pc, const unsigned int Kshared)
{
	unsigned int t1,t2;

	int pcidx1 = (pc-2) & 0xF;
	int pcidx2 = (pc-7) & 0xF;
	int pcidx3 = (pc-15) & 0xF;

	unsigned int inx0 = in[pc];
	unsigned int inx1 = in[pcidx1];
	unsigned int inx2 = in[pcidx2];
	unsigned int inx3 = in[pcidx3];

	unsigned int ssg21 = ssg2_1(inx1);
	unsigned int ssg20 = ssg2_0(inx3);
	unsigned int vxandx = xandx(e, f, g);
	unsigned int bsg21 = bsg2_1(e);
	unsigned int bsg20 = bsg2_0(a);
	unsigned int andorv = andor32(a,b,c);

	in[pc] = ssg21 + inx2 + ssg20 + inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}

__device__
static void sha256_round_body(unsigned int* in, unsigned int* state, unsigned int* const Kshared)
{
    
	unsigned int a = state[0];
	unsigned int b = state[1];
	unsigned int c = state[2];
	unsigned int d = state[3];
	unsigned int e = state[4];
	unsigned int f = state[5];
	unsigned int g = state[6];
	unsigned int h = state[7];

	sha2_step1(a,b,c,d,e,f,g,h,in[ 0], Kshared[ 0]);
	sha2_step1(h,a,b,c,d,e,f,g,in[ 1], Kshared[ 1]);
	sha2_step1(g,h,a,b,c,d,e,f,in[ 2], Kshared[ 2]);
	sha2_step1(f,g,h,a,b,c,d,e,in[ 3], Kshared[ 3]);
	sha2_step1(e,f,g,h,a,b,c,d,in[ 4], Kshared[ 4]);
	sha2_step1(d,e,f,g,h,a,b,c,in[ 5], Kshared[ 5]);
	sha2_step1(c,d,e,f,g,h,a,b,in[ 6], Kshared[ 6]);
	sha2_step1(b,c,d,e,f,g,h,a,in[ 7], Kshared[ 7]);
	sha2_step1(a,b,c,d,e,f,g,h,in[ 8], Kshared[ 8]);
	sha2_step1(h,a,b,c,d,e,f,g,in[ 9], Kshared[ 9]);
	sha2_step1(g,h,a,b,c,d,e,f,in[10], Kshared[10]);
	sha2_step1(f,g,h,a,b,c,d,e,in[11], Kshared[11]);
	sha2_step1(e,f,g,h,a,b,c,d,in[12], Kshared[12]);
	sha2_step1(d,e,f,g,h,a,b,c,in[13], Kshared[13]);
	sha2_step1(c,d,e,f,g,h,a,b,in[14], Kshared[14]);
	sha2_step1(b,c,d,e,f,g,h,a,in[15], Kshared[15]);

	#pragma unroll
	for (int i=0; i<3; i++)
	{
		sha2_step2(a,b,c,d,e,f,g,h,in,0, Kshared[16+16*i]);
		sha2_step2(h,a,b,c,d,e,f,g,in,1, Kshared[17+16*i]);
		sha2_step2(g,h,a,b,c,d,e,f,in,2, Kshared[18+16*i]);
		sha2_step2(f,g,h,a,b,c,d,e,in,3, Kshared[19+16*i]);
		sha2_step2(e,f,g,h,a,b,c,d,in,4, Kshared[20+16*i]);
		sha2_step2(d,e,f,g,h,a,b,c,in,5, Kshared[21+16*i]);
		sha2_step2(c,d,e,f,g,h,a,b,in,6, Kshared[22+16*i]);
		sha2_step2(b,c,d,e,f,g,h,a,in,7, Kshared[23+16*i]);
		sha2_step2(a,b,c,d,e,f,g,h,in,8, Kshared[24+16*i]);
		sha2_step2(h,a,b,c,d,e,f,g,in,9, Kshared[25+16*i]);
		sha2_step2(g,h,a,b,c,d,e,f,in,10,Kshared[26+16*i]);
		sha2_step2(f,g,h,a,b,c,d,e,in,11,Kshared[27+16*i]);
		sha2_step2(e,f,g,h,a,b,c,d,in,12,Kshared[28+16*i]);
		sha2_step2(d,e,f,g,h,a,b,c,in,13,Kshared[29+16*i]);
		sha2_step2(c,d,e,f,g,h,a,b,in,14,Kshared[30+16*i]);
		sha2_step2(b,c,d,e,f,g,h,a,in,15,Kshared[31+16*i]);
	}

	state[0] += a;
	state[1] += b;
	state[2] += c;
	state[3] += d;
	state[4] += e;
	state[5] += f;
	state[6] += g;
	state[7] += h;
}


__device__
static void sha256_round_last(unsigned int* in, unsigned int* state, unsigned int* const Kshared)
{
	unsigned int a = state[0];
	unsigned int b = state[1];
	unsigned int c = state[2];
	unsigned int d = state[3];
	unsigned int e = state[4];
	unsigned int f = state[5];
	unsigned int g = state[6];
	unsigned int h = state[7];

	sha2_step1(a,b,c,d, e,f,g,h, in[ 0], Kshared[ 0]);
	sha2_step1(h,a,b,c, d,e,f,g, in[ 1], Kshared[ 1]);
	sha2_step1(g,h,a,b, c,d,e,f, in[ 2], Kshared[ 2]);
	sha2_step1(f,g,h,a, b,c,d,e, in[ 3], Kshared[ 3]);
	sha2_step1(e,f,g,h, a,b,c,d, in[ 4], Kshared[ 4]);
	sha2_step1(d,e,f,g, h,a,b,c, in[ 5], Kshared[ 5]);
	sha2_step1(c,d,e,f, g,h,a,b, in[ 6], Kshared[ 6]);
	sha2_step1(b,c,d,e, f,g,h,a, in[ 7], Kshared[ 7]);
	sha2_step1(a,b,c,d, e,f,g,h, in[ 8], Kshared[ 8]);
	sha2_step1(h,a,b,c, d,e,f,g, in[ 9], Kshared[ 9]);
	sha2_step1(g,h,a,b, c,d,e,f, in[10], Kshared[10]);
	sha2_step1(f,g,h,a, b,c,d,e, in[11], Kshared[11]);
	sha2_step1(e,f,g,h, a,b,c,d, in[12], Kshared[12]);
	sha2_step1(d,e,f,g, h,a,b,c, in[13], Kshared[13]);
	sha2_step1(c,d,e,f, g,h,a,b, in[14], Kshared[14]);
	sha2_step1(b,c,d,e, f,g,h,a, in[15], Kshared[15]);

	#pragma unroll
	for (int i=0; i<2; i++)
	{
		sha2_step2(a,b,c,d, e,f,g,h, in, 0, Kshared[16+16*i]);
		sha2_step2(h,a,b,c, d,e,f,g, in, 1, Kshared[17+16*i]);
		sha2_step2(g,h,a,b, c,d,e,f, in, 2, Kshared[18+16*i]);
		sha2_step2(f,g,h,a, b,c,d,e, in, 3, Kshared[19+16*i]);
		sha2_step2(e,f,g,h, a,b,c,d, in, 4, Kshared[20+16*i]);
		sha2_step2(d,e,f,g, h,a,b,c, in, 5, Kshared[21+16*i]);
		sha2_step2(c,d,e,f, g,h,a,b, in, 6, Kshared[22+16*i]);
		sha2_step2(b,c,d,e, f,g,h,a, in, 7, Kshared[23+16*i]);
		sha2_step2(a,b,c,d, e,f,g,h, in, 8, Kshared[24+16*i]);
		sha2_step2(h,a,b,c, d,e,f,g, in, 9, Kshared[25+16*i]);
		sha2_step2(g,h,a,b, c,d,e,f, in,10, Kshared[26+16*i]);
		sha2_step2(f,g,h,a, b,c,d,e, in,11, Kshared[27+16*i]);
		sha2_step2(e,f,g,h, a,b,c,d, in,12, Kshared[28+16*i]);
		sha2_step2(d,e,f,g, h,a,b,c, in,13, Kshared[29+16*i]);
		sha2_step2(c,d,e,f, g,h,a,b, in,14, Kshared[30+16*i]);
		sha2_step2(b,c,d,e, f,g,h,a, in,15, Kshared[31+16*i]);
	}

	sha2_step2(a,b,c,d, e,f,g,h, in, 0, Kshared[16+16*2]);
	sha2_step2(h,a,b,c, d,e,f,g, in, 1, Kshared[17+16*2]);
	sha2_step2(g,h,a,b, c,d,e,f, in, 2, Kshared[18+16*2]);
	sha2_step2(f,g,h,a, b,c,d,e, in, 3, Kshared[19+16*2]);
	sha2_step2(e,f,g,h, a,b,c,d, in, 4, Kshared[20+16*2]);
	sha2_step2(d,e,f,g, h,a,b,c, in, 5, Kshared[21+16*2]);
	sha2_step2(c,d,e,f, g,h,a,b, in, 6, Kshared[22+16*2]);
	sha2_step2(b,c,d,e, f,g,h,a, in, 7, Kshared[23+16*2]);
	sha2_step2(a,b,c,d, e,f,g,h, in, 8, Kshared[24+16*2]);
	sha2_step2(h,a,b,c, d,e,f,g, in, 9, Kshared[25+16*2]);
	sha2_step2(g,h,a,b, c,d,e,f, in,10, Kshared[26+16*2]);
	sha2_step2(f,g,h,a, b,c,d,e, in,11, Kshared[27+16*2]);
	sha2_step2(e,f,g,h, a,b,c,d, in,12, Kshared[28+16*2]);
	sha2_step2(d,e,f,g, h,a,b,c, in,13, Kshared[29+16*2]);

	state[6] += g;
	state[7] += h;
}

__device__ __forceinline__
uint64_t cuda_swab32ll(uint64_t x) {
	return MAKE_ULONGLONG(cuda_swab32(_LODWORD(x)), cuda_swab32(_HIDWORD(x)));
}

// 最后一个块的双hash CUDA程序
__global__
/*__launch_bounds__(256,3)*/
void sha256d_gpu_hash_shared(const unsigned int threads, volatile unsigned int *pFinished, const unsigned int startNonce, const unsigned int endNonce, unsigned int *resNonces)
{
	const unsigned int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int nonce = startNonce + thread;
	int iLoopTime = 0;
	volatile __shared__ bool someoneFoundIt;

	if(thread >= threads)
	{
		return;
	}

	if (threadIdx.x == 0) someoneFoundIt = *pFinished;
	__syncthreads();

	// 可能多做一个循环
	int iLoopCount = (endNonce - startNonce + 1) / threads + 1;
	while((!someoneFoundIt) && (iLoopTime < iLoopCount) )
	{
		unsigned int dat[16];
		AS_UINT2(dat) = AS_UINT2(c_dataEnd80);
		dat[ 2] = c_dataEnd80[2];
		dat[ 3] = cuda_swab32(nonce);
		dat[ 4] = 0x80000000;
		dat[15] = 0x280;
		#pragma unroll
		for (int i=5; i<15; i++) dat[i] = 0;

		unsigned int buf[8];
		#pragma unroll
		for (int i=0; i<8; i+=2) AS_UINT2(&buf[i]) = AS_UINT2(&c_midstate76[i]);

		sha256_round_body(dat, buf, c_K);

		// second sha256
		#pragma unroll
		for (int i=0; i<8; i++) dat[i] = buf[i];
		dat[8] = 0x80000000;
		#pragma unroll
		for (int i=9; i<15; i++) dat[i] = 0;
		dat[15] = 0x100;

		#pragma unroll
		for (int i=0; i<8; i++) buf[i] = c_H256[i];

		sha256_round_last(dat, buf, c_K);
//		sha256_round_body(dat, buf, c_K);

		// valid nonces
		uint64_t high = cuda_swab32ll(((uint64_t*)buf)[3]);
		uint64_t *target = (uint64_t *)c_target;
		if (high <= target[0]) {
			unsigned int *pResNonces = (unsigned int *)resNonces;
			resNonces[1] = atomicExch(pResNonces, nonce);
			someoneFoundIt = true;
			*pFinished = true;
		}
		if (threadIdx.x == 0 && *pFinished) someoneFoundIt = true;
		__syncthreads();

		iLoopTime++;
		nonce = startNonce + thread + threads * iLoopTime;
	}
}

__host__
void initSha256d(int deviceId, unsigned int uiBlockNum, unsigned int uiThreadNum)
{
//	cuda_get_arch(deviceId);
	uiBlocksPerGrid = uiBlockNum;
	uiThreadsPerBlock = uiThreadNum;
	hipSetDevice(deviceId);
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	hipMemcpyToSymbol(HIP_SYMBOL(c_K), cpu_K, sizeof(cpu_K), 0, hipMemcpyHostToDevice);
	CUDA_SAFE_CALL(hipMalloc(&d_resNonces[deviceId], 2*sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMalloc(&pFinished, sizeof(unsigned int)));
}

__host__
void freeSha256d(int deviceId)
{
	if (d_resNonces[deviceId]) hipFree(d_resNonces[deviceId]);
	d_resNonces[deviceId] = NULL;
	
	if(pFinished) hipFree((void *)pFinished);
	pFinished = NULL;
}

//  算比特币第一个区块的hash，在CPU上计算，计算结果拷贝到GPU的全局内存（global或constant）
__host__
void hashFirstBlock(unsigned int *pdata, unsigned int *ptarget)
{
	unsigned int __align__(64) in[16], buf[8], end[4];
	for (int i=0;i<16;i++) in[i] = cuda_swab32(pdata[i]);
	for (int i=0;i<8;i++) buf[i] = cpu_H256[i];
	for (int i=0;i<4;i++) end[i] = cuda_swab32(pdata[16+i]);
	sha256_round_body_host(in, buf, cpu_K);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_midstate76), buf, 32, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_dataEnd80),  end, sizeof(end), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_target), &ptarget[6], 8, 0, hipMemcpyHostToDevice));
}

// 计算比特币第二个区块（结束区块）的hash，之后对最终结果再做hash，如果小于目标值，则记录到特定地址（resNonces[1]）
__host__
void hashLastBlock(int deviceId, unsigned int threadNumber, unsigned int startNonce, unsigned int endNonce, unsigned int *resNonces)
{
	CUDA_SAFE_CALL(hipMemset(d_resNonces[deviceId], 0xFF, 2 * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemset((void *)pFinished, 0, sizeof(unsigned int)));
	hipDeviceSynchronize();
	sha256d_gpu_hash_shared <<<uiBlocksPerGrid, uiThreadsPerBlock>>> (threadNumber, pFinished, startNonce, endNonce, d_resNonces[deviceId]);

	hipDeviceSynchronize();
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_resNonces[deviceId], 2 * sizeof(unsigned int), hipMemcpyDeviceToHost));
	if (resNonces[0] == resNonces[1]) {
		resNonces[1] = UINT32_MAX;
	}
}

// 计算单个hash值,无法真正并行
__global__ void sha256SingleRound(unsigned int threads, unsigned int* in, unsigned int* hashValue)
{
    sha256_round_body(in, hashValue, c_K);
    for(int i = 0; i < 8; i++){
		hashValue[i] = cuda_swab32(hashValue[i]);
	}
}

// 单块hash操作
__host__ void hashSingleBlock(unsigned int threads, unsigned int* in, unsigned int* hashValue)
{
	unsigned int *puiDeviceHashValue;
	unsigned int *puiDeviceIn;

	hipMalloc( (void **)&puiDeviceHashValue, 8 * sizeof(int) );
	hipMalloc( (void **)&puiDeviceIn, 16 * sizeof(int) );
	hipMemcpy(puiDeviceHashValue, hashValue, 8 * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(puiDeviceIn, in, 16 * sizeof(int), hipMemcpyHostToDevice );

    sha256SingleRound <<<1, 1>>> (threads, puiDeviceIn, puiDeviceHashValue);
    
    hipMemcpy(hashValue, puiDeviceHashValue, 8 * sizeof(int), hipMemcpyDeviceToHost );
	hipFree(puiDeviceHashValue);
}

__global__ void vecAdd(int *piArray1, int *piArray2, int *piArrayDest, int iLength){
	int tid = threadIdx.x;
	if (tid < iLength){
		piArrayDest[tid] = piArray1[tid] + piArray2[tid];
	}
}

__host__
void vecAddTest(int *piArray1, int *piArray2, int *piArrayDest, int iLength)
{

	int *piArrayDevice1, *piArrayDevice2, *piArrayDeviceDest;
	hipMalloc( (void **)&piArrayDevice1, iLength * sizeof(int) );
    hipMalloc( (void **)&piArrayDevice2, iLength * sizeof(int) );
    hipMalloc( (void **)&piArrayDeviceDest, iLength * sizeof(int) );
	hipMemset( piArrayDeviceDest, 0, iLength * sizeof(int) );
	hipMemcpy(piArrayDevice1, piArray1, iLength * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(piArrayDevice2, piArray2, iLength * sizeof(int), hipMemcpyHostToDevice );

    vecAdd<<<1, iLength>>>(piArrayDevice1, piArrayDevice2, piArrayDeviceDest, iLength);
    hipMemcpy(piArrayDest, piArrayDeviceDest, iLength * sizeof(int), hipMemcpyDeviceToHost );
	hipDeviceSynchronize();
	
	hipFree(piArrayDevice1);
    hipFree(piArrayDevice2);
    hipFree(piArrayDeviceDest);
}
